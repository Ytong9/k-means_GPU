
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
 
int K,N,D;  //聚类的数目，数据量，数据的维数
float **data;  //存放数据
int *in_cluster;  //标记每个点属于哪个聚类
float **cluster_center;  //存放每个聚类的中心点
 
float **array(int m,int n);
void freearray(float **p);
float **loadData(int *k,int *d,int *n);
float getDistance(float avector[],float bvector[],int n);
void cluster();
float getDifference();
void getCenter(int in_cluster[]);
 
int  main()
{
	int i,j,count=0;
	float temp1,temp2;
	data=loadData(&K,&D,&N);
	printf("Data sets:\n");
	for(i=0;i<N;i++)
		for(j=0;j<D;j++){
			printf("%-8.2f",data[i][j]);
			if((j+1)%D==0)    putchar('\n');
		}
		printf("-----------------------------\n");
 
		srand((unsigned int)(time(NULL)));  //随机初始化k个中心点
		for(i=0;i<K;i++)
			for(j=0;j<D;j++)
				cluster_center[i][j]=data[(int)((double)N*rand()/(RAND_MAX+1.0))][j];
 
		cluster();  //用随机k个中心点进行聚类
		temp1=getDifference();  //第一次中心点和所属数据点的距离之和
		count++;
		printf("The difference between data and center is: %.2f\n\n", temp1);
 
		getCenter(in_cluster);
		cluster();  //用新的k个中心点进行第二次聚类
		temp2=getDifference();
		count++;
		printf("The difference between data and center is: %.2f\n\n",temp2);
 
		while(fabs(temp2-temp1)!=0){   //比较前后两次迭代，若不相等继续迭代
			temp1=temp2;
			getCenter(in_cluster);
			cluster();
			temp2=getDifference();
			count++;
			printf("The %dth difference between data and center is: %.2f\n\n",count,temp2);
		}
 
		printf("\nThe total number of cluster is: %d\n",count);  //统计迭代次数
		//system("pause");  //gcc编译需删除 
		return 0;
}
 
 
//动态创建二维数组
float **array(int m,int n)
{
	int i;
	float **p;
	p=(float **)malloc(m*sizeof(float *));
	p[0]=(float *)malloc(m*n*sizeof(float));
	for(i=1;i<m;i++)    p[i]=p[i-1]+n;
	return p;
}
 
//释放二维数组所占用的内存
void freearray(float **p)
{
	free(*p);
	free(p);
}
 
//从data.txt导入数据，要求首行格式：K=聚类数目,D=数据维度,N=数据量
float **loadData(int *k,int *d,int *n)
{
	int i,j; 
	float **arraydata;
	FILE *fp;
	if((fp=fopen("iris.data","r"))==NULL)    fprintf(stderr,"cannot open data!\n");
	if(fscanf(fp,"K=%d D=%d N=%d\n",k,d,n)!=3)        fprintf(stderr,"load error!\n");
	arraydata=array(*n,*d);  //生成数据数组
	cluster_center=array(*k,*d);  //聚类的中心点
	in_cluster=(int *)malloc(*n * sizeof(int));  //每个数据点所属聚类的标志数组
	for(i=0;i<*n;i++)
		for(j=0;j<*d;j++)
			fscanf(fp,"%f,",&arraydata[i][j]);  //读取数据点
	return arraydata;
}
 
//计算欧氏距离
float getDistance(float avector[],float bvector[],int n)
{
	int i;
	float sum=0.0;
	for(i=0;i<n;i++)
		sum+=pow(avector[i]-bvector[i],2);
	return sqrt(sum);
}
 
//把N个数据点聚类，标出每个点属于哪个聚类
void cluster()
{
	int i,j;
	float min;
	float **distance=array(N,K);  //存放每个数据点到每个中心点的距离
	//float distance[N][K];  //也可使用C99变长数组
	FILE *fp2;
	if((fp2=fopen("result.txt","w"))==NULL) {
		printf("File cannot be opened/n");
		exit(0);
	}
    
    clock_t start,end;//生成时间戳
    start = clock();
    
	for(i=0;i<N;++i){
		min=9999.0;
		for(j=0;j<K;++j){
			distance[i][j] = getDistance(data[i],cluster_center[j],D);
			if(distance[i][j]<min){
				min=distance[i][j];
				in_cluster[i]=j;
			}
		}
		printf("data[%d] 属于类-%d\n",i,in_cluster[i]);
		fprintf(fp2,"%d \n",in_cluster[i]);//写入文件
	}
    end = clock();
	printf("------------time=%fms-----------------\n",(double)(end-start)/1000);//CLK_TCK =1000
	free(distance);
}
 
//计算所有聚类的中心点与其数据点的距离之和
float getDifference()
{
	int i,j;
	float sum=0.0;
	for(i=0;i<K;++i){
		for(j=0;j<N;++j){
			if(i==in_cluster[j])
				sum+=getDistance(data[j],cluster_center[i],D);
		}
	}
	return sum;
}
 
//计算每个聚类的中心点
void getCenter(int in_cluster[])
{
	float **sum=array(K,D);  //存放每个聚类中心点
	//float sum[K][D];  //也可使用C99变长数组
	int i,j,q,count;
	for(i=0;i<K;i++)
		for(j=0;j<D;j++)
			sum[i][j]=0.0;
	for(i=0;i<K;i++){
		count=0;  //统计属于某个聚类内的所有数据点
		for(j=0;j<N;j++){
			if(i==in_cluster[j]){
				for(q=0;q<D;q++)
					sum[i][q]+=data[j][q];  //计算所属聚类的所有数据点的相应维数之和
				count++;
			}
		}
		for(q=0;q<D;q++)
			cluster_center[i][q]=sum[i][q]/count;
	}
	printf("The new center of cluster is:\n");
	for(i = 0; i < K; i++)
		for(q=0;q<D;q++){
			printf("%-8.2f",cluster_center[i][q]);
			if((q+1)%D==0)    putchar('\n');
		}
		free(sum);
}
