#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"

 
int K,N,D;  //聚类的数目，数据量，数据的维数
float **data;  //存放数据
int *in_cluster;  //标记每个点属于哪个聚类
float **cluster_center;  //存放每个聚类的中心点
 
float **array(int m,int n);
void freearray(float **p);
float **loadData(int *k,int *d,int *n);
float getDistance(float avector[],float bvector[],int n);
void cluster();
float getDifference();
void getCenter(int in_cluster[]);
void cluster1();


int  main()
{
	int i,j,count=0;
	float temp1,temp2;
	data=loadData(&K,&D,&N);
	printf("Data sets:\n");
	for(i=0;i<N;i++)
		for(j=0;j<D;j++){
			printf("%-8.2f",data[i][j]);
			if((j+1)%D==0)    putchar('\n');
		}
		printf("-----------------------------\n");
 
		srand((unsigned int)(time(NULL)));  //随机初始化k个中心点
		for(i=0;i<K;i++)
			for(j=0;j<D;j++)
				cluster_center[i][j]=data[(int)((double)N*rand()/(RAND_MAX+1.0))][j];
 
		cluster();  //用随机k个中心点进行聚类
		temp1=getDifference();  //第一次中心点和所属数据点的距离之和
		count++;
		printf("The difference between data and center is: %.2f\n\n", temp1);
 
		getCenter(in_cluster);
		
        cluster();  //用新的k个中心点进行第二次聚类
    
		temp2=getDifference();
		count++;
		printf("The difference between data and center is: %.2f\n\n",temp2);
 
		while(fabs(temp2-temp1)!=0){   //比较前后两次迭代，若不相等继续迭代
			temp1=temp2;
			getCenter(in_cluster);
			cluster();
			temp2=getDifference();
			count++;
			printf("The %dth difference between data and center is: %.2f\n\n",count,temp2);
		}

		printf("\nThe total number of cluster is: %d\n",count);  //统计迭代次数
		//system("pause");  //gcc编译需删除 
		return 0;
       
}
 
 
//动态创建二维数组
float **array(int m,int n)
{
	int i;
	float **p;
	p=(float **)malloc(m*sizeof(float *));
	p[0]=(float *)malloc(m*n*sizeof(float));
	for(i=1;i<m;i++)    p[i]=p[i-1]+n;
	return p;
}
 
//释放二维数组所占用的内存
void freearray(float **p)
{
	free(*p);
	free(p);
}
 
//从data.txt导入数据，要求首行格式：K=聚类数目,D=数据维度,N=数据量
float **loadData(int *k,int *d,int *n)
{
	int i,j; 
	float **arraydata;
	FILE *fp;
	if((fp=fopen("iris.data","r"))==NULL)    fprintf(stderr,"cannot open data!\n");
	if(fscanf(fp,"K=%d D=%d N=%d\n",k,d,n)!=3)        fprintf(stderr,"load error!\n");
	arraydata=array(*n,*d);  //生成数据数组
	cluster_center=array(*k,*d);  //聚类的中心点
	in_cluster=(int *)malloc(*n * sizeof(int));  //每个数据点所属聚类的标志数组
	for(i=0;i<*n;i++)
		for(j=0;j<*d;j++)
			fscanf(fp,"%f,",&arraydata[i][j]);  //读取数据点
	return arraydata;
}
 
//计算欧氏距离
float getDistance(float avector[],float bvector[],int n)
{
	int i;
	float sum=0.0;
	for(i=0;i<n;i++)
		sum+=pow(avector[i]-bvector[i],2);
	return sqrt(sum);
}
 

__device__ float GetDistance(float avector[],float bvector[],int n)
{
	int i;
	float sum=0.0;
	for(i=0;i<n;i++)
    {
		sum+=(avector[i]-bvector[i])*(avector[i]-bvector[i]);
    }
	return sqrt(sum);
}

__global__ void ComputeDistance(float* distance,float **data_2d,float **cc_2d,int d)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
        //printf("第%d与聚类%d的距离是",i,j);
        distance[i*blockDim.x+j] = GetDistance(data_2d[i],cc_2d[j],d);
        //printf("属于聚类%d\n",in_cluster_h[i]);
}


//把N个数据点聚类，标出每个点属于哪个聚类
__host__ void cluster()
{
    float *distance;
    hipMalloc((void**)&distance,sizeof(float)*N*K);//在GPU中申请distance的内存
    
	//float **distance=array(N,K);  //存放每个数据点到每个中心点的距离
	//float distance[N][K];  //也可使用C99变长数组
    float **data_2d,**cc_2d;
    float **host_data_2d = array(N,K);
    float **host_cc_2d = array(K,D);
    for(int i = 0;i < N; i++)
    {
        //float *host_data_1d = (float*)malloc(sizeof(float)*K);
        float *data_1d;
        hipMalloc((void**)&data_1d,sizeof(float)*D);
        hipMemcpy(data_1d,data[i],sizeof(float)*D,hipMemcpyHostToDevice);
        host_data_2d[i] = data_1d;
    }
    hipMalloc((void**)&data_2d,sizeof(float*)*N);
    hipMemcpy(data_2d,host_data_2d,sizeof(float*)*N,hipMemcpyHostToDevice);
    //将二维数组data传入GPU
    for(int i = 0;i < K; i++)
    {
        //float *host_data_1d = (float*)malloc(sizeof(float)*K);
        
        float *cc_1d;
        hipMalloc((void**)&cc_1d,sizeof(float)*D);
        hipMemcpy(cc_1d,cluster_center[i],sizeof(float)*D,hipMemcpyHostToDevice);
        host_cc_2d[i] = cc_1d;
    }
    hipMalloc((void**)&cc_2d,sizeof(float*)*K);
    hipMemcpy(cc_2d,host_cc_2d,sizeof(float*)*K,hipMemcpyHostToDevice);
    

    //float tmp=9999.0;
    //cudaMemcpyToSymbol(mi, &tmp, sizeof(float)); //在cuda中设置全局常量
    

    
    dim3 GridSize(N,1,1);
    dim3 BlockSize(K,1,1);
    
    clock_t start,end;//生成时间戳
    start = clock();

    ComputeDistance<<<GridSize,BlockSize>>>(distance,data_2d,cc_2d,D);
    hipDeviceSynchronize();
    
    end = clock();
    
    float* dis = (float*)malloc(sizeof(float)*N*K);
    hipMemcpy(dis,distance,sizeof(int)*N,hipMemcpyDeviceToHost);
    
    
    FILE *fp2;
    if((fp2=fopen("result.txt","w"))==NULL) 
    {
		printf("File cannot be opened/n");
		exit(0);
	}
    
	for(int i=0;i<N;++i){
		float min=9999.0;
		for(int j=0;j<K;++j){
			dis[i*K+j] = getDistance(data[i],cluster_center[j],D);
			if(dis[i*K+j]<min){
				min=dis[i*K+j];
				in_cluster[i]=j;
			}
		}
		printf("data[%d] 属于类-%d\n",i,in_cluster[i]);
		fprintf(fp2,"%d \n",in_cluster[i]);//写入文件
	}
	printf("------------time=%fms-----------------\n",(double)(end-start)/1000);//CLK_TCK =1000
	hipFree(distance);
    hipFree(data_2d);
    hipFree(cc_2d);
}
 
//计算所有聚类的中心点与其数据点的距离之和
float getDifference()
{
	int i,j;
	float sum=0.0;
	for(i=0;i<K;++i){
		for(j=0;j<N;++j){
			if(i==in_cluster[j])
				sum+=getDistance(data[j],cluster_center[i],D);
		}
	}
	return sum;
}
 
//计算每个聚类的中心点
void getCenter(int in_cluster[])
{
	float **sum=array(K,D);  //存放每个聚类中心点
	//float sum[K][D];  //也可使用C99变长数组
	int i,j,q,count;
	for(i=0;i<K;i++)
		for(j=0;j<D;j++)
			sum[i][j]=0.0;
	for(i=0;i<K;i++){
		count=0;  //统计属于某个聚类内的所有数据点
		for(j=0;j<N;j++){
			if(i==in_cluster[j]){
				for(q=0;q<D;q++)
					sum[i][q]+=data[j][q];  //计算所属聚类的所有数据点的相应维数之和
				count++;
			}
		}
		for(q=0;q<D;q++)
			cluster_center[i][q]=sum[i][q]/count;
	}
	printf("The new center of cluster is:\n");
	for(i = 0; i < K; i++)
		for(q=0;q<D;q++){
			printf("%-8.2f",cluster_center[i][q]);
			if((q+1)%D==0)    putchar('\n');
		}
		free(sum);
}
